#include "hip/hip_runtime.h"
#include "model_utils.cuh"

hipError_t Conv2dWithCuda(struct tensor* input_tensor, struct tensor* kernel, int kernel_size, int stride, int num_channels, int num_filters, struct tensor* output_tensor) {
	// This function takes the struct input_image, the array kernel and perform the convolution using the GPU.
	// The resulting image will stored inside the struct 'output_image'.

	int memsize_input_tensor = input_tensor->col * input_tensor->row * input_tensor->depth * sizeof(float);
	int memsize_kernel = kernel_size * kernel_size * num_channels * sizeof(float);
	int memsize_output_tensor = output_tensor->col * output_tensor->row * output_tensor->depth * sizeof(float);

	// Declaration of the input_array, kernel and output_array and move them to the GPU.
	float* dev_input_data;
	hipMalloc((void**)&dev_input_data, memsize_input_tensor);
	hipMemcpy(dev_input_data, input_tensor->data, memsize_input_tensor, hipMemcpyHostToDevice);

	float* dev_kernel;
	hipMalloc((void**)&dev_kernel, memsize_kernel);
	hipMemcpy(dev_kernel, kernel->data, memsize_kernel, hipMemcpyHostToDevice);

	float* dev_output_data;
	hipMalloc((void**)&dev_output_data, memsize_output_tensor);
	// No need to copy the output tensor to device before computation

	// Define CudaKernel settings.
	dim3 threadInBlock(1, 1, 8); // Adjust to suitable block size
	dim3 numBlocks;
	numBlocks.x = (input_tensor->col + threadInBlock.x - 1) / threadInBlock.x;
	numBlocks.y = (input_tensor->row + threadInBlock.y - 1) / threadInBlock.y;
	int memsize_shared_memory = threadInBlock.z * sizeof(float);

	// Get the starting time.
	hipError_t cudaStatus;

	// Launch the cuda kernel that performs the convolution.
	convolution_parallel << <numBlocks, threadInBlock, memsize_shared_memory >> > (dev_input_data, input_tensor->row, input_tensor->col, input_tensor->depth, dev_kernel, kernel_size, stride, dev_output_data);

	// Compute the elapsed time in ms.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "convGPU launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// Move the output array from Device back to host.
	hipMemcpy(output_tensor->data, dev_output_data, memsize_output_tensor, hipMemcpyDeviceToHost);

	hipFree(dev_input_data);
	hipFree(dev_kernel);
	hipFree(dev_output_data);

	return cudaStatus;
}

__global__ void convolution_parallel(float* input_tensor, int nrow, int ncol, int nchannels, float* kernel, int kernel_size, int stride, float* output_tensor) {
    // This function defines the kernel execute in every GPU's thread.
    // In the GPU version, we don't need the outer for loop to iterate over the all image. But, each thread operates on a single sub-image.

    extern __shared__ float tmp_conv_channels[];

    // Compute the padding size
    int pad = kernel_size / 2;

    // Get the row, col, and channel of the image the thread is pointing on considering the stride factor.
    int row = (threadIdx.y + blockIdx.y * blockDim.y) * stride;
    int col = (threadIdx.x + blockIdx.x * blockDim.x) * stride;
    int tid = threadIdx.z;	
	int channel = tid;

    // Ensure the pixel is inside a valid region of the image.
    if ((row < nrow) && (col < ncol) && (tid < nchannels)) {
        float result = 0.0f;

		// Padding (make it more efficient)
		int start_row = 0;
		int start_col = 0;
		int end_row = kernel_size;
		int end_col = kernel_size;

		if (row < pad) {
			start_row = pad - row;
		}
		if (col < pad) {
			start_col = pad - col;
		}
		if (row > (nrow - pad - 1)) {
			end_row = pad + (nrow - row);
		}
		if (col > (ncol - pad - 1)) {
			end_col = pad + (ncol - col);
		}

		// Convolution
		while (channel < nchannels) {
			for (int i = start_row; i < end_row; i++) {
				for (int j = start_col; j < end_col; j++) {
					int img_row = row + i - pad;
					int img_col = col + j - pad;
					int img_idx = channel * nrow * ncol + img_row * ncol + img_col;
					int kernel_idx = channel * kernel_size * kernel_size + i * kernel_size + j;
					result += kernel[kernel_idx] * input_tensor[img_idx];
				}
			}
			channel += blockDim.z;
		}
        tmp_conv_channels[tid] = result;
    }

    // Synchronize to ensure all threads have written to shared memory
    __syncthreads();

	// Reduction algorithm to sum results across all channels using the shared memory
	for (int s = blockDim.z / 2; s > 0; s >>= 1) {
		if (tid < s) {
			tmp_conv_channels[tid] += tmp_conv_channels[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) {
		int output_image_idx = (blockIdx.y * blockDim.y + threadIdx.y) * (ncol / stride) + (blockIdx.x * blockDim.x + threadIdx.x);
		output_tensor[output_image_idx] = tmp_conv_channels[0];
	}
}

// Riparametrizzazione convoluzione
	/*
	if ((row >= pad && row < (nrow - pad)) && (col >= pad && col < (ncol - pad))) {
		int start_idx = -kernel_size / 2;
		int result = 0;
		int input_image_idx = (nrow * ncol) * channel;
		int kernel_idx = (kernel_size * kernel_size) * channel;

		for (int i = start_idx; i <= kernel_size / 2; i++) {
			for (int j = start_idx; j <= kernel_size / 2; j++) {
				result += (kernel[kernel_idx] * input_tensor[input_image_idx + ncol * (row + i) + (col + j)]);
				kernel_idx++;
			}
		}
		tmp_conv_channels[channel] = result;
	}
	*/